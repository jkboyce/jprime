#include "hip/hip_runtime.h"
//
// CoordinatorCuda.cu
//
// Core graph search routines, implemented as iterative functions that are drop-
// in replacements for recursive versions in GenLoopsRecursive.cc. These
// routines are by far the most performance-critical portions of jprime.
//
// Copyright (C) 1998-2025 Jack Boyce, <jboyce@gmail.com>
//
// This file is distributed under the MIT License.
//

#include "Coordinator.h"

#include <iostream>
#include <vector>


__global__ void helloCUDA(int* a, int* b, int* c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}


bool Coordinator::run_cuda() {
  std::vector<int> a, b, c;

  for (int i = 0; i < 100; ++i) {
    a.push_back(i);
    b.push_back(i);
  }
  c.assign(a.size(), 0);

  helloCUDA<<<1, 100>>>(a.data(), b.data(), c.data());
  hipDeviceSynchronize();

  for (int i = 0; i < c.size(); ++i) {
    std::cout << c[i] << "\n";
  }

  return true;
}
