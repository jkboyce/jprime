#include "hip/hip_runtime.h"
//
// CoordinatorCUDA.cu
//
// Routines for executing the search on a CUDA-enabled GPU. This file should
// be compiled with `nvcc`, part of the CUDA Toolkit.
//
// Copyright (C) 1998-2025 Jack Boyce, <jboyce@gmail.com>
//
// This file is distributed under the MIT License.
//

#include "Coordinator.h"
#include "Graph.h"

#include "CoordinatorCUDA.cuh"

#include <iostream>
#include <vector>
#include <format>
#include <sstream>
#include <algorithm>
#include <stdexcept>
#include <cassert>

#include <hip/hip_runtime.h>



//------------------------------------------------------------------------------
// GPU memory layout
//------------------------------------------------------------------------------

// GPU constant memory
//
// Every NVIDIA GPU from capability 5.0 through 12.0 has 64 KB of constant
// memory. This is where we place the juggling graph data.

__device__ __constant__ statenum_t graphmatrix_d[65536 / sizeof(statenum_t)];


// GPU global memory

__device__ uint8_t maxoutdegree_d;
__device__ uint8_t unused_d;
__device__ uint16_t numstates_d;
__device__ uint32_t pattern_buffer_size_d;
__device__ uint32_t pattern_index_d = 0;


//------------------------------------------------------------------------------
// GPU kernels
//------------------------------------------------------------------------------

__global__ void cuda_gen_loops_normal(statenum_t* const patterns_d,
        WorkerInfo* const wi_d, WorkAssignmentCell* const wa_d,
        const unsigned n_min, const unsigned n_max, const unsigned steps,
        const bool report) {
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (wi_d[id].done) {
    return;
  }

  // set up register variables
  statenum_t st_state = wi_d[id].start_state;
  int pos = wi_d[id].pos;
  uint64_t nnodes = wi_d[id].nnodes;
  const uint8_t outdegree = maxoutdegree_d;

  // set up shared memory
  //
  // unused[] arrays for 32 threads are stored in (numstates_d + 1) instances
  // of ThreadStorageUsed, each of which is 32 uint32s
  //
  // WorkAssignmentCell[] arrays for 32 threads are stored in (n_max)
  // instances of ThreadStorageWorkCell, each of which is 64 uint32s

  extern __shared__ uint32_t shared[];
  ThreadStorageUsed* used = (ThreadStorageUsed*)
      &shared[(threadIdx.x / 32) * 32 * (numstates_d + 1) + (threadIdx.x % 32)];
  ThreadStorageWorkCell* workcell = (ThreadStorageWorkCell*)
      &shared[
          ((blockDim.x + 31) / 32) * 32 * (numstates_d + 1) +
          (threadIdx.x / 32) * 64 * n_max + (threadIdx.x % 32)
      ];

  /*
  int shared_memory_size_bytes =
      ((blockDim.x + 31) / 32) * 128 * (numstates_d + 1) +
      ((blockDim.x + 31) / 32) * 256 * n_max;
  printf("shared memory size (device) = %d bytes\n", shared_memory_size_bytes);
  */

  // initialize workcell[] array
  for (int i = 0; i < n_max; ++i) {
    workcell[i].col = wa_d[id * n_max + i].col;
    workcell[i].col_limit = wa_d[id * n_max + i].col_limit;
    workcell[i].from_state = wa_d[id * n_max + i].from_state;
    workcell[i].count = wa_d[id * n_max + i].count;
  }

  // initialize used[] array
  for (int i = 0; i <= numstates_d; ++i) {
    used[i].used = 0;
  }
  for (int i = 1; i <= pos; ++i) {
    used[workcell[i].from_state].used = 1;
  }

  ThreadStorageWorkCell* ss = &workcell[pos];

  for (unsigned step = 0; ; ++step) {
    if (ss->col == ss->col_limit) {
      // beat is finished, go back to previous one
      used[ss->from_state].used = 0;
      ++nnodes;

      if (pos == 0) {
        if (st_state == wi_d[id].end_state) {
          wi_d[id].done = 1;
          break;
        }
        ++st_state;
        ss->col = 0;
        ss->col_limit = outdegree;
        ss->from_state = st_state;
        continue;
      } else {
        --pos;
        --ss;
        ++ss->col;
        continue;
      }
    }

    const statenum_t to_state = graphmatrix_d[(ss->from_state - 1) *
          outdegree + ss->col];

    if (to_state == 0) {
      // beat is finished, go back to previous one
      used[ss->from_state].used = 0;
      ++nnodes;

      if (pos == 0) {
        if (st_state == wi_d[id].end_state) {
          wi_d[id].done = 1;
          break;
        }
        ++st_state;
        ss->col = 0;
        ss->col_limit = outdegree;
        ss->from_state = st_state;
        continue;
      } else {
        --pos;
        --ss;
        ++ss->col;
        continue;
      }
    }
    
    if (to_state == st_state) {
      // found a valid pattern
      if (report && pos + 1 >= n_min) {
        const uint32_t idx = atomicAdd(&pattern_index_d, 1);
        if (idx < pattern_buffer_size_d) {
          for (int j = 0; j <= pos; ++j) {
            patterns_d[idx * n_max + j] = workcell[j].from_state;
          }
          if (pos + 1 < n_max) {
            patterns_d[idx * n_max + pos + 1] = 0;
          }
        }
      }
      ++ss->count;
      ++ss->col;
      continue;
    }

    if (to_state < st_state) {
      ++ss->col;
      continue;
    }

    if (used[to_state].used) {
      ++ss->col;
      continue;
    }

    if (pos + 1 == n_max) {
      ++ss->col;
      continue;
    }

    // current throw is valid, so advance to next beat

    if (step > steps)
      break;

    ++pos;
    ++ss;
    ss->col = 0;
    ss->col_limit = outdegree;
    ss->from_state = to_state;
    used[to_state].used = 1;
  }

  wi_d[id].start_state = st_state;
  wi_d[id].pos = pos;
  wi_d[id].nnodes = nnodes;

  // save workcell[] array
  for (int i = 0; i < n_max; ++i) {
    wa_d[id * n_max + i].col = workcell[i].col;
    wa_d[id * n_max + i].col_limit = workcell[i].col_limit;
    wa_d[id * n_max + i].from_state = workcell[i].from_state;
    wa_d[id * n_max + i].count = workcell[i].count;
  }
}


__global__ void cuda_gen_loops_normal_global(statenum_t* const patterns_d,
        WorkerInfo* const wi_d, WorkAssignmentCell* const wa_d,
        const unsigned n_min, const unsigned n_max, const unsigned steps,
        const bool report) {
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (wi_d[id].done) {
    return;
  }

  // set up register variables
  statenum_t st_state = wi_d[id].start_state;
  int pos = wi_d[id].pos;
  uint64_t nnodes = wi_d[id].nnodes;
  const uint8_t outdegree = maxoutdegree_d;

  // set up shared memory
  //
  // unused[] arrays for 32 threads are stored in (numstates_d + 1) instances
  // of ThreadStorageUsed, each of which is 32 uint32s

  extern __shared__ uint32_t shared[];
  ThreadStorageUsed* used = (ThreadStorageUsed*)
      &shared[(threadIdx.x / 32) * 32 * (numstates_d + 1) + (threadIdx.x % 32)];

  // initialize used[] array
  for (int i = 0; i <= numstates_d; ++i) {
    used[i].used = 0;
  }
  for (int i = 1; i <= pos; ++i) {
    used[wa_d[id * n_max + i].from_state].used = 1;
  }

  WorkAssignmentCell* ss = &wa_d[id * n_max + pos];

  for (unsigned step = 0; ; ++step) {
    if (ss->col == ss->col_limit) {
      // beat is finished, go back to previous one
      used[ss->from_state].used = 0;
      ++nnodes;

      if (pos == 0) {
        if (st_state == wi_d[id].end_state) {
          wi_d[id].done = 1;
          break;
        }
        ++st_state;
        ss->col = 0;
        ss->col_limit = outdegree;
        ss->from_state = st_state;
        continue;
      } else {
        --pos;
        --ss;
        ++ss->col;
        continue;
      }
    }

    const statenum_t to_state = graphmatrix_d[(ss->from_state - 1) *
          outdegree + ss->col];

    if (to_state == 0) {
      // beat is finished, go back to previous one
      used[ss->from_state].used = 0;
      ++nnodes;

      if (pos == 0) {
        if (st_state == wi_d[id].end_state) {
          wi_d[id].done = 1;
          break;
        }
        ++st_state;
        ss->col = 0;
        ss->col_limit = outdegree;
        ss->from_state = st_state;
        continue;
      } else {
        --pos;
        --ss;
        ++ss->col;
        continue;
      }
    }
    
    if (to_state == st_state) {
      // found a valid pattern
      if (report && pos + 1 >= n_min) {
        const uint32_t idx = atomicAdd(&pattern_index_d, 1);
        if (idx < pattern_buffer_size_d) {
          for (int j = 0; j <= pos; ++j) {
            patterns_d[idx * n_max + j] = wa_d[id * n_max + j].from_state;
          }
          if (pos + 1 < n_max) {
            patterns_d[idx * n_max + pos + 1] = 0;
          }
        }
      }
      ++ss->count;
      ++ss->col;
      continue;
    }

    if (to_state < st_state) {
      ++ss->col;
      continue;
    }

    if (used[to_state].used) {
      ++ss->col;
      continue;
    }

    if (pos + 1 == n_max) {
      ++ss->col;
      continue;
    }

    // current throw is valid, so advance to next beat

    if (step > steps)
      break;

    ++pos;
    ++ss;
    ss->col = 0;
    ss->col_limit = outdegree;
    ss->from_state = to_state;
    used[to_state].used = 1;
  }

  wi_d[id].start_state = st_state;
  wi_d[id].pos = pos;
  wi_d[id].nnodes = nnodes;
}


//------------------------------------------------------------------------------
// Benchmarks
//------------------------------------------------------------------------------

/*
20 blocks, 32 threads/block:
jprime 3 9 -cuda -count
30513071763 patterns in range (30513071763 seen, 141933075458 nodes)
runtime = 238.0548 sec (596.2M nodes/sec, 0.0 % util, 14803 splits)

1 block, 32 threads/block:
jprime 3 8 -cuda -count
11906414 patterns in range (11906414 seen, 49962563 nodes)
runtime = 1.7509 sec (28.5M nodes/sec, 0.0 % util, 306 splits)

1 block, 64 threads/block:
jprime 3 8 -cuda -count
11906414 patterns in range (11906414 seen, 49962563 nodes)
runtime = 1.0728 sec (46.6M nodes/sec, 0.0 % util, 533 splits)

1 block, 96 threads/block:
jprime 3 8 -cuda -count
11906414 patterns in range (11906414 seen, 49962563 nodes)
runtime = 0.8166 sec (61.2M nodes/sec, 0.0 % util, 765 splits)

2 blocks, 32 threads/block:
jprime 3 8 -cuda -count
11906414 patterns in range (11906414 seen, 49962563 nodes)
runtime = 1.0862 sec (46.0M nodes/sec, 0.0 % util, 533 splits)

50 blocks, 96 threads/block:
jprime 3 9 -cuda -count
shared memory size = 89472 bytes
steps per kernel call = 200000
30513071763 patterns in range (30513071763 seen, 141933075458 nodes)
runtime = 36.7760 sec (3859.4M nodes/sec, 0.0 % util, 81821 splits)
89.9 sec (20000 steps, 56 x 96)
33.7 sec (200000 steps, 56 x 96)
32.1 sec (300000 steps, 56 x 96) *
53.1 sec (300000 steps, global memory, 56 x 96)
43.1 sec (300000 steps, global memory, 56 x 288) *
--> 61.0728 sec on 10 CPU cores


*/

//------------------------------------------------------------------------------
// Execution entry point
//------------------------------------------------------------------------------

void Coordinator::run_cuda() {
  const unsigned num_blocks = 56;
  const unsigned num_threadsperblock = 32 * 3;
  num_workers = num_blocks * num_threadsperblock;
  unsigned num_steps = 300000;
  pattern_buffer_size = 100000;
  
  // 1. Initialization

  (void)initialize_cuda_device();
  Graph graph = build_and_reduce_graph();
  jpout << "Execution parameters:\n"
        << "  num_blocks = " << num_blocks
        << "\n  num_threadsperblock = " << num_threadsperblock
        << "\n  num_workers = " << num_workers
        << "\n  steps per kernel call = " << num_steps << std::endl;
  CudaAlgorithm alg = select_CUDA_search_algorithm(graph);
  check_memory_limits(graph, alg, num_threadsperblock);
  configure_cuda_shared_memory();

  allocate_gpu_memory();
  copy_graph_to_gpu(graph, alg);
  copy_static_vars_to_gpu(graph);

  std::vector<WorkerInfo> wi_h(num_workers);
  std::vector<WorkAssignmentCell> wa_h(num_workers * n_max);

  load_initial_work_assignments(graph, wi_h, wa_h);

  // 2. Main Loop

  while (true) {
    copy_worker_data_to_gpu(wi_h, wa_h);
    launch_cuda_kernel(alg, num_blocks, num_threadsperblock, num_steps);
    copy_worker_data_from_gpu(wi_h, wa_h);

    process_worker_results(graph, wi_h, wa_h);
    process_pattern_buffer(pb_d, graph, pattern_buffer_size);

    bool any_done = false;
    bool all_done = true;
    for (const auto &wi : wi_h) {
      if (wi.done) {
        any_done = true;
      } else {
        all_done = false;
      }
    }
    
    // Termination condition
    if (Coordinator::stopping || all_done)
      break;

    if (any_done) {
      assign_new_jobs(graph, wi_h, wa_h);
    }
  }

  // 3. Cleanup

  cleanup_gpu_memory();
  gather_unfinished_work_assignments(graph, wi_h, wa_h);
}

//------------------------------------------------------------------------------
// Setup
//------------------------------------------------------------------------------

// Initialize CUDA device and check properties.

hipDeviceProp_t Coordinator::initialize_cuda_device() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  jpout << "Device Number: " << 0
        << "\n  device name: " << prop.name
        << "\n  multiprocessor count: " << prop.multiProcessorCount
        << "\n  total global memory (bytes): " << prop.totalGlobalMem
        << "\n  total constant memory (bytes): " << prop.totalConstMem
        << "\n  shared memory per block (bytes): " << prop.sharedMemPerBlock
        << "\n  shared memory per block, maximum opt-in (bytes): "
        << prop.sharedMemPerBlockOptin << std::endl;

  return prop;
}

// Build and reduce the juggling graph.

Graph Coordinator::build_and_reduce_graph() {
  Graph graph = {
      config.b,
      config.h,
      config.xarray,
      config.graphmode == SearchConfig::GraphMode::SINGLE_PERIOD_GRAPH
                     ? config.n_min : 0
  };
  graph.build_graph();
  // TODO: call customize_graph() here
  graph.reduce_graph();
  return graph;
}

// choose a search algorithm to use

CudaAlgorithm Coordinator::select_CUDA_search_algorithm(const Graph& graph)
      const {
  unsigned max_possible = (config.mode == SearchConfig::RunMode::SUPER_SEARCH)
      ? graph.superprime_period_bound(config.shiftlimit)
      : graph.prime_period_bound();

  CudaAlgorithm alg = CudaAlgorithm::NONE;

  if (config.mode == SearchConfig::RunMode::NORMAL_SEARCH) {
    if (config.graphmode == SearchConfig::GraphMode::FULL_GRAPH &&
        static_cast<double>(config.n_min) >
        0.66 * static_cast<double>(max_possible)) {
      // the overhead of marking is only worth it for long-period patterns
      alg = CudaAlgorithm::NORMAL_MARKING;
    } else if (config.countflag) {
      alg = CudaAlgorithm::NORMAL;
    } else {
      alg = CudaAlgorithm::NORMAL;
    }
  } else if (config.mode == SearchConfig::RunMode::SUPER_SEARCH) {
    if (config.shiftlimit == 0) {
      alg = CudaAlgorithm::SUPER0;
    } else {
      alg = CudaAlgorithm::SUPER;
    }
  }

  if (config.verboseflag) {
    jpout << "selected algorithm " << cuda_algs[static_cast<int>(alg)]
          << std::endl;
  }

  return alg;
}

// Check if the graph and work data fit in GPU memory.

void Coordinator::check_memory_limits(const Graph& graph, CudaAlgorithm alg,
        unsigned num_threadsperblock) {
  const unsigned graphcols =
      (alg == CudaAlgorithm::NORMAL || alg == CudaAlgorithm::NORMAL_GLOBAL)
      ? graph.maxoutdegree : graph.maxoutdegree + 1;
  const size_t graph_buffer_size =
      graph.numstates * graphcols * sizeof(statenum_t);

  if (graph_buffer_size > sizeof(graphmatrix_d)) {
    throw std::runtime_error("CUDA error: Juggling graph too large");
  }

  if (alg == CudaAlgorithm::NORMAL || alg == CudaAlgorithm::NORMAL_MARKING) {
    // put WorkAssignmentCells in shared memory
    shared_memory_size = ((num_threadsperblock + 31) / 32) * (
        128 * (graph.numstates + 1) +  // used[]
        256 * n_max                    // WorkAssignentCell[]
    );
  } else if (alg == CudaAlgorithm::NORMAL_GLOBAL) {
    // leave WorkAssignmentCells in global memory
    shared_memory_size = ((num_threadsperblock + 31) / 32) * (
        128 * (graph.numstates + 1)    // used[]
    );
  }

  jpout << "  shared memory req'd = " << shared_memory_size << " bytes"
        << std::endl;


  if (shared_memory_size > 99 * 1024) {
    // TODO: This comparison should be based on queried device properties
    throw std::runtime_error("CUDA error: Not enough shared memory");
  }
}

// Set up CUDA shared memory configuration.

void Coordinator::configure_cuda_shared_memory() {
  hipFuncSetAttribute(reinterpret_cast<const void*>(cuda_gen_loops_normal),
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size);
  hipFuncSetAttribute(reinterpret_cast<const void*>(cuda_gen_loops_normal_global),
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size);
}

// Allocate GPU memory for patterns, WorkerInfo, and WorkAssignmentCells.

void Coordinator::allocate_gpu_memory() {
  throw_on_cuda_error(
      hipMalloc(&pb_d, sizeof(statenum_t) * n_max * pattern_buffer_size),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMalloc(&wi_d, sizeof(WorkerInfo) * num_workers),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMalloc(&wa_d, sizeof(WorkAssignmentCell) * num_workers * n_max),
      __FILE__, __LINE__);
}

// Copy graph data to GPU constant memory.

void Coordinator::copy_graph_to_gpu(const Graph& graph, CudaAlgorithm alg) {
  const unsigned graphcols =
      (alg == CudaAlgorithm::NORMAL || alg == CudaAlgorithm::NORMAL_GLOBAL)
      ? graph.maxoutdegree : graph.maxoutdegree + 1;
  const size_t graph_buffer_size =
      graph.numstates * graphcols * sizeof(statenum_t);

  std::vector<statenum_t> graph_buffer(graph_buffer_size, 0);

  for (unsigned i = 1; i <= graph.numstates; ++i) {
    for (unsigned j = 0; j < graph.outdegree.at(i); ++j) {
      graph_buffer.at((i - 1) * graphcols + j) = graph.outmatrix.at(i).at(j);
    }
    if (alg == CudaAlgorithm::NORMAL_MARKING) {
      graph_buffer.at((i - 1) * graphcols + graph.maxoutdegree) =
          graph.upstream_state(i);
    }
    if (alg == CudaAlgorithm::SUPER0 || alg == CudaAlgorithm::SUPER) {
      graph_buffer.at((i - 1) * graphcols + graph.maxoutdegree) =
          graph.cyclenum.at(i);
    }
  }

  throw_on_cuda_error(
      hipMemcpyToSymbol(HIP_SYMBOL(graphmatrix_d), graph_buffer.data(),
                         sizeof(statenum_t) * graph_buffer.size()),
      __FILE__, __LINE__);
}

// Copy static global variables to GPU global memory.

void Coordinator::copy_static_vars_to_gpu(const Graph& graph) {
  uint8_t maxoutdegree_h = static_cast<uint8_t>(graph.maxoutdegree);
  uint16_t numstates_h = static_cast<uint16_t>(graph.numstates);
  uint32_t pattern_buffer_size_h = pattern_buffer_size;
  uint32_t pattern_index_h = 0;
  throw_on_cuda_error(
      hipMemcpyToSymbol(HIP_SYMBOL(maxoutdegree_d), &maxoutdegree_h, sizeof(uint8_t)),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMemcpyToSymbol(HIP_SYMBOL(numstates_d), &numstates_h, sizeof(uint16_t)),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMemcpyToSymbol(HIP_SYMBOL(pattern_buffer_size_d), &pattern_buffer_size_h,
                         sizeof(uint32_t)),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMemcpyToSymbol(HIP_SYMBOL(pattern_index_d), &pattern_index_h, sizeof(uint32_t)),
      __FILE__, __LINE__);
}

//------------------------------------------------------------------------------
// Main loop
//------------------------------------------------------------------------------

// Copy worker data to the GPU.

void Coordinator::copy_worker_data_to_gpu(std::vector<WorkerInfo>& wi_h,
    std::vector<WorkAssignmentCell>& wa_h) {
  throw_on_cuda_error(
      hipMemcpy(wi_d, wi_h.data(), sizeof(WorkerInfo) * num_workers,
          hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMemcpy(wa_d, wa_h.data(),
          sizeof(WorkAssignmentCell) * num_workers * n_max,
          hipMemcpyHostToDevice),
      __FILE__, __LINE__);
}

// Launch the appropriate CUDA kernel.

void Coordinator::launch_cuda_kernel(CudaAlgorithm alg, unsigned num_blocks,
      unsigned num_threadsperblock, unsigned num_steps) {
  switch (alg) {
    case CudaAlgorithm::NORMAL:
      cuda_gen_loops_normal
        <<<num_blocks, num_threadsperblock, shared_memory_size>>>
        (pb_d, wi_d, wa_d, config.n_min, n_max, num_steps, !config.countflag);
      break;
    case CudaAlgorithm::NORMAL_GLOBAL:
      cuda_gen_loops_normal_global
        <<<num_blocks, num_threadsperblock, shared_memory_size>>>
        (pb_d, wi_d, wa_d, config.n_min, n_max, num_steps, !config.countflag);
      break;
    default:
      throw std::runtime_error("CUDA error: algorithm not implemented");
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(std::format("CUDA Error: {}",
        hipGetErrorString(err)));
  }

  hipDeviceSynchronize();
}

// Copy worker data from the GPU.

void Coordinator::copy_worker_data_from_gpu(std::vector<WorkerInfo>& wi_h,
    std::vector<WorkAssignmentCell>& wa_h) {
  throw_on_cuda_error(
      hipMemcpy(wi_h.data(), wi_d, sizeof(WorkerInfo) * num_workers,
          hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  throw_on_cuda_error(
      hipMemcpy(wa_h.data(), wa_d,
          sizeof(WorkAssignmentCell) * num_workers * n_max,
          hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

// Process worker results and handle pattern buffer.

void Coordinator::process_worker_results(const Graph& graph,
      std::vector<WorkerInfo>& wi_h, std::vector<WorkAssignmentCell>& wa_h) {
  int num_working = 0;
  int num_idle = 0;

  for (int id = 0; id < num_workers; ++id) {
    if (wi_h.at(id).done) {
      ++num_idle;
    } else {
      ++num_working;
    }

    MessageW2C msg;
    msg.worker_id = id;
    msg.count.assign(n_max + 1, 0);
    for (unsigned j = 0; j < n_max; ++j) {
      msg.count.at(j + 1) = wa_h.at(id * n_max + j).count;
      wa_h.at(id * n_max + j).count = 0;
    }
    msg.nnodes = wi_h.at(id).nnodes;
    wi_h.at(id).nnodes = 0;
    record_data_from_message(msg);
  }
}

// Process the pattern buffer, copying any patterns to `context` and printing
// them to the console if needed. Then clear the buffer.
//
// In the event of a buffer overflow, throw a `std::runtime_error` exception
// with a relevant error message.

void Coordinator::process_pattern_buffer(statenum_t* const pb_d,
    const Graph& graph, const uint32_t pattern_buffer_size) {
  // get the number of patterns in the buffer
  uint32_t pattern_count;
  throw_on_cuda_error(
    hipMemcpyFromSymbol(&pattern_count, HIP_SYMBOL(pattern_index_d), sizeof(uint32_t)),
    __FILE__, __LINE__
  );

  if (pattern_count == 0) {
    return;
  } else if (pattern_count > pattern_buffer_size) {
    throw std::runtime_error("CUDA error: pattern buffer overflow");
  }
    
  // copy pattern data to host
  std::vector<statenum_t> patterns_h(n_max * pattern_count);
  throw_on_cuda_error(
    hipMemcpy(patterns_h.data(), pb_d, sizeof(statenum_t) * n_max *
        pattern_count, hipMemcpyDeviceToHost),
    __FILE__, __LINE__
  );

  // work out each pattern's throw values from the list of state numbers
  // traversed, and process them

  std::vector<int> pattern_throws(n_max + 1);

  for (int i = 0; i < pattern_count; ++i) {
    const statenum_t start_state = patterns_h.at(i * n_max);
    statenum_t from_state = start_state;
    unsigned period = 0;

    for (int j = 0; j < n_max; ++j) {
      statenum_t to_state = (j == n_max - 1) ? start_state :
                              patterns_h.at(i * n_max + j + 1);
      if (to_state == 0) {
        to_state = start_state;
      }
    
      int throwval = -1;
      for (unsigned k = 0; k < graph.outdegree.at(from_state); ++k) {
        if (graph.outmatrix.at(from_state).at(k) == to_state) {
          throwval = graph.outthrowval.at(from_state).at(k);
          break;
        }
      }
      if (throwval == -1) {
        // diagnostic information in case of a problem
        std::cerr << "pattern count = " << pattern_count << '\n';
        std::cerr << "i = " << i << '\n';
        std::cerr << "j = " << j << '\n';
        for (unsigned k = 0; k < n_max; ++k) {
          statenum_t st = patterns_h.at(i * n_max + k);
          if (st == 0)
            break;
          std::cerr << "state(" << k << ") = " << graph.state.at(st) << '\n';
        }
        std::cerr << "from_state = " << from_state << " (" << graph.state.at(from_state)
                  << ")\n";
        std::cerr << "to_state = " << to_state << '\n';
        std::cerr << "outdegree(from_state) = " << graph.outdegree.at(from_state) << '\n';
        for (unsigned k = 0; k < graph.outdegree.at(from_state); ++k) {
          std::cerr << "outmatrix(from_state)[" << k << "] = "
                    << graph.outmatrix.at(from_state).at(k)
                    << " (" << graph.state.at(graph.outmatrix.at(from_state).at(k)) << ")\n";
        }
        throw std::runtime_error("CUDA error: invalid pattern");
      }
      pattern_throws.at(j) = throwval;

      ++period;
      if (to_state == start_state) {
        pattern_throws.at(j + 1) = -1;  // signals end of the pattern
        break;
      }
      from_state = to_state;
    }

    MessageW2C msg;
    msg.worker_id = 0;
    msg.pattern = pattern_output_format(config, pattern_throws, start_state);
    msg.period = period;
    process_search_result(msg);
  }

  // reset the pattern buffer index

  uint32_t pattern_index_h = 0;
  throw_on_cuda_error(
    hipMemcpyToSymbol(HIP_SYMBOL(pattern_index_d), &pattern_index_h, sizeof(uint32_t)),
    __FILE__, __LINE__
  );
}


//------------------------------------------------------------------------------
// Cleanup
//------------------------------------------------------------------------------

// Clean up GPU memory.

void Coordinator::cleanup_gpu_memory() {
  hipFree(pb_d);
  hipFree(wi_d);
  hipFree(wa_d);
}

// Gather unfinished work assignments.

void Coordinator::gather_unfinished_work_assignments(const Graph& graph,
    std::vector<WorkerInfo>& wi_h, std::vector<WorkAssignmentCell>& wa_h) {
  for (unsigned id = 0; id < num_workers; ++id) {
    if (!wi_h.at(id).done) {
      WorkAssignment wa = read_work_assignment(id, wi_h, wa_h, graph);
      context.assignments.push_back(wa);
    }
  }
}

//------------------------------------------------------------------------------
// Manage work assignments
//------------------------------------------------------------------------------

// Load initial work assignments.

void Coordinator::load_initial_work_assignments(const Graph& graph,
      std::vector<WorkerInfo>& wi_h, std::vector<WorkAssignmentCell>& wa_h)  {
  for (int id = 0; id < num_workers; ++id) {
    if (context.assignments.size() > 0) {
      WorkAssignment wa = context.assignments.front();
      context.assignments.pop_front();
      load_work_assignment(id, wa, wi_h, wa_h, graph);

      if (config.verboseflag) {
        erase_status_output();
        jpout << std::format("worker {} given work:\n  ", id)
              << wa << std::endl;
        print_status_output();
      }
    } else {
      wi_h.at(id).done = 1;
    }
  }
}

// Load a work assignment into a worker's slot in the `WorkerInfo` and
// `WorkAssignmentCell` arrays.

void Coordinator::load_work_assignment(const unsigned id,
    const WorkAssignment& wa, std::vector<WorkerInfo>& wi_h,
    std::vector<WorkAssignmentCell>& wa_h, const Graph& graph) {
  unsigned start_state = wa.start_state;
  unsigned end_state = wa.end_state;
  if (start_state == 0) {
    start_state = (config.groundmode ==
        SearchConfig::GroundMode::EXCITED_SEARCH ? 2 : 1);
  }
  if (end_state == 0) {
    end_state = (config.groundmode ==
        SearchConfig::GroundMode::GROUND_SEARCH ? 1 : graph.numstates);
  }

  wi_h.at(id).start_state = start_state;
  wi_h.at(id).end_state = end_state;
  wi_h.at(id).pos = wa.partial_pattern.size();
  wi_h.at(id).nnodes = 0;
  wi_h.at(id).done = 0;

  // set up WorkAssignmentCells

  for (unsigned i = 0; i < n_max; ++i) {
    wa_h.at(id * n_max + i).count = 0;
  }

  // default if `wa.partial_pattern` is empty
  wa_h.at(id * n_max).col = 0;
  wa_h.at(id * n_max).col_limit = static_cast<uint8_t>(graph.maxoutdegree);
  wa_h.at(id * n_max).from_state = start_state;

  unsigned from_state = start_state;

  for (unsigned i = 0; i < wa.partial_pattern.size(); ++i) {
    const unsigned tv = wa.partial_pattern.at(i);
    unsigned to_state = 0;

    for (unsigned j = 0; j < graph.outdegree.at(from_state); ++j) {
      if (graph.outthrowval.at(from_state).at(j) != tv)
        continue;

      to_state = graph.outmatrix.at(from_state).at(j);

      wa_h.at(id * n_max + i).col = static_cast<uint8_t>(j);
      wa_h.at(id * n_max + i).col_limit = (i < wa.root_pos ?
          static_cast<uint8_t>(j + 1) :
          static_cast<uint8_t>(graph.maxoutdegree));

      wa_h.at(id * n_max + i + 1).col = 0;
      wa_h.at(id * n_max + i + 1).col_limit =
          static_cast<uint8_t>(graph.maxoutdegree);
      wa_h.at(id * n_max + i + 1).from_state = to_state;
      break;
    }
    if (to_state == 0) {
      std::cerr << "problem loading work assignment:\n   "
                << wa
                << "\nat position " << i
                << std::endl;

      throw std::runtime_error("CUDA error: problem loading work assignment");
    }

    from_state = to_state;
  }

  // fix `col` and `col_limit` at position `root_pos`
  if (wa.root_throwval_options.size() > 0) {
    statenum_t from_state = wa_h.at(id * n_max + wa.root_pos).from_state;
    unsigned col = graph.maxoutdegree;
    unsigned col_limit = 0;

    for (unsigned i = 0; i < graph.outdegree.at(from_state); ++i) {
      const unsigned tv = graph.outthrowval.at(from_state).at(i);
      auto it = std::find(wa.root_throwval_options.begin(),
          wa.root_throwval_options.end(), tv);
      if (it != wa.root_throwval_options.end()) {
        col = std::min(i, col);
        col_limit = i + 1;
      }
      if (wa.root_pos < wa.partial_pattern.size() &&
          tv == wa.partial_pattern.at(wa.root_pos)) {
        col = std::min(i, col);
        col_limit = i + 1;
      }
    }

    wa_h.at(id * n_max + wa.root_pos).col = col;
    wa_h.at(id * n_max + wa.root_pos).col_limit = col_limit;
  }

  /*
  if (config.statusflag) {
    worker_options_left_start.at(id).resize(0);
    worker_options_left_last.at(id).resize(0);
    worker_longest_start.at(id) = 0;
    worker_longest_last.at(id) = 0;
  }
  */
}

// Read out the current work assignment for worker `id`.

WorkAssignment Coordinator::read_work_assignment(unsigned id,
    std::vector<WorkerInfo>& wi_h, std::vector<WorkAssignmentCell>& wa_h,
    const Graph& graph) {
  WorkAssignment wa;

  wa.start_state = wi_h.at(id).start_state;
  wa.end_state = wi_h.at(id).end_state;

  bool root_pos_found = false;

  for (unsigned i = 0; i <= wi_h.at(id).pos; ++i) {
    const unsigned from_state = wa_h.at(id * n_max + i).from_state;
    unsigned col = wa_h.at(id * n_max + i).col;
    const unsigned col_limit = std::min(graph.outdegree.at(from_state),
                  static_cast<unsigned>(wa_h.at(id * n_max + i).col_limit));

    wa.partial_pattern.push_back(graph.outthrowval.at(from_state).at(col));

    if (col < col_limit - 1 && !root_pos_found) {
      wa.root_pos = i;
      root_pos_found = true;

      ++col;
      while (col < col_limit) {
        wa.root_throwval_options.push_back(
            graph.outthrowval.at(from_state).at(col));
        ++col;
      }
    }
  }

  return wa;
}

// Assign new jobs to idle workers

void Coordinator::assign_new_jobs(const Graph& graph,
    std::vector<WorkerInfo>& wi_h, std::vector<WorkAssignmentCell>& wa_h) {

  // sort the running work assignments to find the best ones to split
  std::vector<WorkAssignmentLine> sorted_assignments;
  for (unsigned id = 0; id < num_workers; ++id) {
    if (!wi_h.at(id).done) {
      WorkAssignment wa = read_work_assignment(id, wi_h, wa_h, graph);
      sorted_assignments.push_back({id, wa});
    }
  }

  // compare function returns true if the first argument appears before the
  // second in a strict weak ordering, and false otherwise
  std::sort(sorted_assignments.begin(), sorted_assignments.end(),
      [](WorkAssignmentLine wal1, WorkAssignmentLine wal2) {
        return work_assignment_compare(wal1.wa, wal2.wa);
      }
  );

  unsigned index = 0;
  for (unsigned id = 0; id < num_workers; ++id) {
    if (!wi_h.at(id).done)
      continue;

    if (context.assignments.size() > 0) {
      WorkAssignment wa = context.assignments.front();
      context.assignments.pop_front();
      load_work_assignment(id, wa, wi_h, wa_h, graph);

      if (config.verboseflag) {
        jpout << std::format("worker {} given work:\n   ", id)
              << wa << '\n';
      }
      continue;
    }
  
    // split one of the running jobs and give it to worker `id`

    bool success = false;
    while (!success) {
      if (index == sorted_assignments.size())
        break;

      WorkAssignmentLine& wal = sorted_assignments.at(index);
      WorkAssignment wa = wal.wa;

      if (config.verboseflag) {
        jpout << std::format("worker {} went idle\n", id)
              << std::format("stealing from worker {}\n", wal.id)
              << "work before:\n" << wa << '\n';
      }

      try {
        WorkAssignment wa2 = wa.split(graph, config.split_alg);
        load_work_assignment(wal.id, wa, wi_h, wa_h, graph);
        load_work_assignment(id, wa2, wi_h, wa_h, graph);
        
        if (config.verboseflag) {
          jpout << "work after:\n" << wa << '\n'
                << std::format("new work for worker {}:\n", id)
                << wa2 << '\n';
        }

        // Avoid double counting nodes: Each of the "prefix" nodes up to and
        // including `wa2.root_pos` will be reported twice: by the worker that
        // was running, and by the worker `id` who just got job `wa2`.
        if (wa.start_state == wa2.start_state) {
          wi_h.at(id).nnodes -= (wa2.root_pos + 1);
        }
        ++context.splits_total;
        success = true;
      } catch (const std::invalid_argument& ia) {
      }
      ++index;
    }

    if (index == sorted_assignments.size())
      break;
  }
}

//------------------------------------------------------------------------------
// Helper methods
//------------------------------------------------------------------------------

// Handle CUDA errors by throwing a `std::runtime_error` exception with a
// relevant error message.

void Coordinator::throw_on_cuda_error(hipError_t code, const char *file,
      int line) {
  if (code != hipSuccess) {
    std::stringstream ss;
    ss << "CUDA error: " << hipGetErrorString(code) << " in file "
      << file << " at line " << line;
    throw std::runtime_error(ss.str());
  }
}
